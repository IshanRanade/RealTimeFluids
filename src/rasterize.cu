#include "hip/hip_runtime.h"
/**
 * @file      rasterize.cu
 * @brief     CUDA-accelerated rasterization pipeline.
 * @authors   Salaar Kohari (Skeleton code: Yining Karl Li, Kai Ninomiya, Shuai Shao)
 * @date      2012-2018
 * @copyright University of Pennsylvania & STUDENT
 */

#include <cmath>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/random.h>
#include <util/checkCUDAError.h>
#include <util/tiny_gltf_loader.h>
#include "rasterizeTools.h"
#include "rasterize.h"
#include <glm/gtc/quaternion.hpp>
#include <glm/gtc/matrix_transform.hpp>

namespace {

	typedef unsigned short VertexIndex;
	typedef glm::vec3 VertexAttributePosition;
	typedef glm::vec3 VertexAttributeNormal;
	typedef glm::vec2 VertexAttributeTexcoord;
	typedef unsigned char TextureData;

	typedef unsigned char BufferByte;

	enum PrimitiveType{
		Point = 1,
		Line = 2,
		Triangle = 3
	};

	struct VertexOut {
		glm::vec4 pos;

		// TODO: add new attributes to your VertexOut
		// The attributes listed below might be useful, 
		// but always feel free to modify on your own

		glm::vec3 eyePos;	// eye space position used for shading
		glm::vec3 eyeNor;	// eye space normal used for shading, cuz normal will go wrong after perspective transformation
		glm::vec3 color;
		glm::vec2 texcoord0;
		TextureData* dev_diffuseTex = NULL;
		int texWidth, texHeight;
		// ...
	};

	struct Primitive {
		PrimitiveType primitiveType = Triangle;	// C++ 11 init
		VertexOut v[3];
	};

	struct Fragment {
		glm::vec3 color;

		// TODO: add new attributes to your Fragment
		// The attributes listed below might be useful, 
		// but always feel free to modify on your own

		// glm::vec3 eyePos;	// eye space position used for shading
		glm::vec3 eyeNor;
		// VertexAttributeTexcoord texcoord0;
		// TextureData* dev_diffuseTex;
		// ...
	};

	struct PrimitiveDevBufPointers {
		int primitiveMode;	//from tinygltfloader macro
		PrimitiveType primitiveType;
		int numPrimitives;
		int numIndices;
		int numVertices;

		// Vertex In, const after loaded
		VertexIndex* dev_indices;
		VertexAttributePosition* dev_position;
		VertexAttributeNormal* dev_normal;
		VertexAttributeTexcoord* dev_texcoord0;

		// Materials, add more attributes when needed
		TextureData* dev_diffuseTex;
		int diffuseTexWidth;
		int diffuseTexHeight;
		// TextureData* dev_specularTex;
		// TextureData* dev_normalTex;
		// ...

		// Vertex Out, vertex used for rasterization, this is changing every frame
		VertexOut* dev_verticesOut;

		// TODO: add more attributes when needed
	};

}

static std::map<std::string, std::vector<PrimitiveDevBufPointers>> mesh2PrimitivesMap;


static int width = 0;
static int height = 0;

static int totalNumPrimitives = 0;
static Primitive *dev_primitives = NULL;
static Fragment *dev_fragmentBuffer = NULL;
static glm::vec3 *dev_framebuffer = NULL;

static int * dev_depth = NULL;	// you might need this buffer when doing depth test

static int* mutex = NULL;

/**
 * Kernel that writes the image to the OpenGL PBO directly.
 */
__global__ 
void sendImageToPBO(uchar4 *pbo, int w, int h, glm::vec3 *image) {
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    int index = x + (y * w);

    if (x < w && y < h) {
        glm::vec3 color;
        color.x = glm::clamp(image[index].x, 0.0f, 1.0f) * 255.0;
        color.y = glm::clamp(image[index].y, 0.0f, 1.0f) * 255.0;
        color.z = glm::clamp(image[index].z, 0.0f, 1.0f) * 255.0;
        // Each thread writes one pixel location in the texture (textel)
        pbo[index].w = 0;
        pbo[index].x = color.x;
        pbo[index].y = color.y;
        pbo[index].z = color.z;
    }
}

/** 
* Writes fragment colors to the framebuffer
*/
__global__
void render(const int w, const int h, Fragment *fragmentBuffer, glm::vec3 *framebuffer) {
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    int index = x + (y * w);

    if (x < w && y < h) {
		// TODO: add your fragment shader code here
		const glm::vec3 lightVec = glm::vec3(-0.5774, -0.5774f, 0.5774f);
		glm::vec3 frameColor;
		glm::vec3 eyeNor;

		for (int i = 0; i < SSAA; i++) {
			for (int j = 0; j < SSAA; j++) {
				int aaindex = (x * SSAA) + i + (((y * SSAA) + j) * w * SSAA);
				frameColor += fragmentBuffer[aaindex].color;
				eyeNor += fragmentBuffer[aaindex].eyeNor;
			}
		}
		frameColor /= (SSAA * SSAA);
		eyeNor /= (SSAA * SSAA);

		float lambert = glm::clamp(glm::dot(eyeNor, lightVec), 0.06f, 1.0f);
		framebuffer[index] = frameColor * lambert;
    }
}

/**
 * Called once at the beginning of the program to allocate memory.
 */
void rasterizeInit(int w, int h) {
    width = w * SSAA;
    height = h * SSAA;
	hipFree(dev_fragmentBuffer);
	hipMalloc(&dev_fragmentBuffer, width * height * sizeof(Fragment));
	hipMemset(dev_fragmentBuffer, 0, width * height * sizeof(Fragment));
    hipFree(dev_framebuffer);
    hipMalloc(&dev_framebuffer,   w * h * sizeof(glm::vec3));
    hipMemset(dev_framebuffer, 0, w * h * sizeof(glm::vec3));
    
	hipFree(dev_depth);
	hipMalloc(&dev_depth, width * height * sizeof(int));

	hipMalloc((void **)&mutex, width * height * sizeof(int));
	hipMemset(mutex, 0, width * height * sizeof(int));

	checkCUDAError("rasterizeInit");
}

__global__
void initDepth(int w, int h, int * depth)
{
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;

	if (x < w && y < h)
	{
		int index = x + (y * w);
		depth[index] = INT_MAX;
	}
}


/**
* kern function with support for stride to sometimes replace hipMemcpy
* One thread is responsible for copying one component
*/
__global__ 
void _deviceBufferCopy(int N, BufferByte* dev_dst, const BufferByte* dev_src, int n, int byteStride, int byteOffset, int componentTypeByteSize) {
	
	// Attribute (vec3 position)
	// component (3 * float)
	// byte (4 * byte)

	// id of component
	int i = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (i < N) {
		int count = i / n;
		int offset = i - count * n;	// which component of the attribute

		for (int j = 0; j < componentTypeByteSize; j++) {
			
			dev_dst[count * componentTypeByteSize * n 
				+ offset * componentTypeByteSize 
				+ j]

				= 

			dev_src[byteOffset 
				+ count * (byteStride == 0 ? componentTypeByteSize * n : byteStride) 
				+ offset * componentTypeByteSize 
				+ j];
		}
	}
	

}

__global__
void _nodeMatrixTransform(
	int numVertices,
	VertexAttributePosition* position,
	VertexAttributeNormal* normal,
	glm::mat4 MV, glm::mat3 MV_normal) {

	// vertex id
	int vid = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (vid < numVertices) {
		position[vid] = glm::vec3(MV * glm::vec4(position[vid], 1.0f));
		normal[vid] = glm::normalize(MV_normal * normal[vid]);
	}
}

glm::mat4 getMatrixFromNodeMatrixVector(const tinygltf::Node & n) {
	
	glm::mat4 curMatrix(1.0);

	const std::vector<double> &m = n.matrix;
	if (m.size() > 0) {
		// matrix, copy it

		for (int i = 0; i < 4; i++) {
			for (int j = 0; j < 4; j++) {
				curMatrix[i][j] = (float)m.at(4 * i + j);
			}
		}
	} else {
		// no matrix, use rotation, scale, translation

		if (n.translation.size() > 0) {
			curMatrix[3][0] = n.translation[0];
			curMatrix[3][1] = n.translation[1];
			curMatrix[3][2] = n.translation[2];
		}

		if (n.rotation.size() > 0) {
			glm::mat4 R;
			glm::quat q;
			q[0] = n.rotation[0];
			q[1] = n.rotation[1];
			q[2] = n.rotation[2];

			R = glm::mat4_cast(q);
			curMatrix = curMatrix * R;
		}

		if (n.scale.size() > 0) {
			curMatrix = curMatrix * glm::scale(glm::vec3(n.scale[0], n.scale[1], n.scale[2]));
		}
	}

	return curMatrix;
}

void traverseNode (
	std::map<std::string, glm::mat4> & n2m,
	const tinygltf::Scene & scene,
	const std::string & nodeString,
	const glm::mat4 & parentMatrix
	) 
{
	const tinygltf::Node & n = scene.nodes.at(nodeString);
	glm::mat4 M = parentMatrix * getMatrixFromNodeMatrixVector(n);
	n2m.insert(std::pair<std::string, glm::mat4>(nodeString, M));

	auto it = n.children.begin();
	auto itEnd = n.children.end();

	for (; it != itEnd; ++it) {
		traverseNode(n2m, scene, *it, M);
	}
}

void rasterizeSetBuffers(const tinygltf::Scene & scene) {

	totalNumPrimitives = 0;

	std::map<std::string, BufferByte*> bufferViewDevPointers;

	// 1. copy all `bufferViews` to device memory
	{
		std::map<std::string, tinygltf::BufferView>::const_iterator it(
			scene.bufferViews.begin());
		std::map<std::string, tinygltf::BufferView>::const_iterator itEnd(
			scene.bufferViews.end());

		for (; it != itEnd; it++) {
			const std::string key = it->first;
			const tinygltf::BufferView &bufferView = it->second;
			if (bufferView.target == 0) {
				continue; // Unsupported bufferView.
			}

			const tinygltf::Buffer &buffer = scene.buffers.at(bufferView.buffer);

			BufferByte* dev_bufferView;
			hipMalloc(&dev_bufferView, bufferView.byteLength);
			hipMemcpy(dev_bufferView, &buffer.data.front() + bufferView.byteOffset, bufferView.byteLength, hipMemcpyHostToDevice);

			checkCUDAError("Set BufferView Device Mem");

			bufferViewDevPointers.insert(std::make_pair(key, dev_bufferView));

		}
	}



	// 2. for each mesh: 
	//		for each primitive: 
	//			build device buffer of indices, materail, and each attributes
	//			and store these pointers in a map
	{

		std::map<std::string, glm::mat4> nodeString2Matrix;
		auto rootNodeNamesList = scene.scenes.at(scene.defaultScene);

		{
			auto it = rootNodeNamesList.begin();
			auto itEnd = rootNodeNamesList.end();
			for (; it != itEnd; ++it) {
				traverseNode(nodeString2Matrix, scene, *it, glm::mat4(1.0f));
			}
		}


		// parse through node to access mesh

		auto itNode = nodeString2Matrix.begin();
		auto itEndNode = nodeString2Matrix.end();
		for (; itNode != itEndNode; ++itNode) {

			const tinygltf::Node & N = scene.nodes.at(itNode->first);
			const glm::mat4 & matrix = itNode->second;
			const glm::mat3 & matrixNormal = glm::transpose(glm::inverse(glm::mat3(matrix)));

			auto itMeshName = N.meshes.begin();
			auto itEndMeshName = N.meshes.end();

			for (; itMeshName != itEndMeshName; ++itMeshName) {

				const tinygltf::Mesh & mesh = scene.meshes.at(*itMeshName);

				auto res = mesh2PrimitivesMap.insert(std::pair<std::string, std::vector<PrimitiveDevBufPointers>>(mesh.name, std::vector<PrimitiveDevBufPointers>()));
				std::vector<PrimitiveDevBufPointers> & primitiveVector = (res.first)->second;

				// for each primitive
				for (size_t i = 0; i < mesh.primitives.size(); i++) {
					const tinygltf::Primitive &primitive = mesh.primitives[i];

					if (primitive.indices.empty())
						return;

					// TODO: add new attributes for your PrimitiveDevBufPointers when you add new attributes
					VertexIndex* dev_indices = NULL;
					VertexAttributePosition* dev_position = NULL;
					VertexAttributeNormal* dev_normal = NULL;
					VertexAttributeTexcoord* dev_texcoord0 = NULL;

					// ----------Indices-------------

					const tinygltf::Accessor &indexAccessor = scene.accessors.at(primitive.indices);
					const tinygltf::BufferView &bufferView = scene.bufferViews.at(indexAccessor.bufferView);
					BufferByte* dev_bufferView = bufferViewDevPointers.at(indexAccessor.bufferView);

					// assume type is SCALAR for indices
					int n = 1;
					int numIndices = indexAccessor.count;
					int componentTypeByteSize = sizeof(VertexIndex);
					int byteLength = numIndices * n * componentTypeByteSize;

					dim3 numThreadsPerBlock(128);
					dim3 numBlocks((numIndices + numThreadsPerBlock.x - 1) / numThreadsPerBlock.x);
					hipMalloc(&dev_indices, byteLength);
					_deviceBufferCopy << <numBlocks, numThreadsPerBlock >> > (
						numIndices,
						(BufferByte*)dev_indices,
						dev_bufferView,
						n,
						indexAccessor.byteStride,
						indexAccessor.byteOffset,
						componentTypeByteSize);


					checkCUDAError("Set Index Buffer");


					// ---------Primitive Info-------

					// Warning: LINE_STRIP is not supported in tinygltfloader
					int numPrimitives;
					PrimitiveType primitiveType;
					switch (primitive.mode) {
					case TINYGLTF_MODE_TRIANGLES:
						primitiveType = PrimitiveType::Triangle;
						numPrimitives = numIndices / 3;
						break;
					case TINYGLTF_MODE_TRIANGLE_STRIP:
						primitiveType = PrimitiveType::Triangle;
						numPrimitives = numIndices - 2;
						break;
					case TINYGLTF_MODE_TRIANGLE_FAN:
						primitiveType = PrimitiveType::Triangle;
						numPrimitives = numIndices - 2;
						break;
					case TINYGLTF_MODE_LINE:
						primitiveType = PrimitiveType::Line;
						numPrimitives = numIndices / 2;
						break;
					case TINYGLTF_MODE_LINE_LOOP:
						primitiveType = PrimitiveType::Line;
						numPrimitives = numIndices + 1;
						break;
					case TINYGLTF_MODE_POINTS:
						primitiveType = PrimitiveType::Point;
						numPrimitives = numIndices;
						break;
					default:
						// output error
						break;
					};


					// ----------Attributes-------------

					auto it(primitive.attributes.begin());
					auto itEnd(primitive.attributes.end());

					int numVertices = 0;
					// for each attribute
					for (; it != itEnd; it++) {
						const tinygltf::Accessor &accessor = scene.accessors.at(it->second);
						const tinygltf::BufferView &bufferView = scene.bufferViews.at(accessor.bufferView);

						int n = 1;
						if (accessor.type == TINYGLTF_TYPE_SCALAR) {
							n = 1;
						}
						else if (accessor.type == TINYGLTF_TYPE_VEC2) {
							n = 2;
						}
						else if (accessor.type == TINYGLTF_TYPE_VEC3) {
							n = 3;
						}
						else if (accessor.type == TINYGLTF_TYPE_VEC4) {
							n = 4;
						}

						BufferByte * dev_bufferView = bufferViewDevPointers.at(accessor.bufferView);
						BufferByte ** dev_attribute = NULL;

						numVertices = accessor.count;
						int componentTypeByteSize;

						// Note: since the type of our attribute array (dev_position) is static (float32)
						// We assume the glTF model attribute type are 5126(FLOAT) here

						if (it->first.compare("POSITION") == 0) {
							componentTypeByteSize = sizeof(VertexAttributePosition) / n;
							dev_attribute = (BufferByte**)&dev_position;
						}
						else if (it->first.compare("NORMAL") == 0) {
							componentTypeByteSize = sizeof(VertexAttributeNormal) / n;
							dev_attribute = (BufferByte**)&dev_normal;
						}
						else if (it->first.compare("TEXCOORD_0") == 0) {
							componentTypeByteSize = sizeof(VertexAttributeTexcoord) / n;
							dev_attribute = (BufferByte**)&dev_texcoord0;
						}

						std::cout << accessor.bufferView << "  -  " << it->second << "  -  " << it->first << '\n';

						dim3 numThreadsPerBlock(128);
						dim3 numBlocks((n * numVertices + numThreadsPerBlock.x - 1) / numThreadsPerBlock.x);
						int byteLength = numVertices * n * componentTypeByteSize;
						hipMalloc(dev_attribute, byteLength);

						_deviceBufferCopy << <numBlocks, numThreadsPerBlock >> > (
							n * numVertices,
							*dev_attribute,
							dev_bufferView,
							n,
							accessor.byteStride,
							accessor.byteOffset,
							componentTypeByteSize);

						std::string msg = "Set Attribute Buffer: " + it->first;
						checkCUDAError(msg.c_str());
					}

					// malloc for VertexOut
					VertexOut* dev_vertexOut;
					hipMalloc(&dev_vertexOut, numVertices * sizeof(VertexOut));
					checkCUDAError("Malloc VertexOut Buffer");

					// ----------Materials-------------

					// You can only worry about this part once you started to 
					// implement textures for your rasterizer
					TextureData* dev_diffuseTex = NULL;
					int diffuseTexWidth = 0;
					int diffuseTexHeight = 0;
					if (!primitive.material.empty()) {
						const tinygltf::Material &mat = scene.materials.at(primitive.material);
						printf("material.name = %s\n", mat.name.c_str());

						if (mat.values.find("diffuse") != mat.values.end()) {
							std::string diffuseTexName = mat.values.at("diffuse").string_value;
							if (scene.textures.find(diffuseTexName) != scene.textures.end()) {
								const tinygltf::Texture &tex = scene.textures.at(diffuseTexName);
								if (scene.images.find(tex.source) != scene.images.end()) {
									const tinygltf::Image &image = scene.images.at(tex.source);

									size_t s = image.image.size() * sizeof(TextureData);
									hipMalloc(&dev_diffuseTex, s);
									hipMemcpy(dev_diffuseTex, &image.image.at(0), s, hipMemcpyHostToDevice);
									
									diffuseTexWidth = image.width;
									diffuseTexHeight = image.height;

									checkCUDAError("Set Texture Image data");
								}
							}
						}

						// TODO: write your code for other materials
						// You may have to take a look at tinygltfloader
						// You can also use the above code loading diffuse material as a start point 
					}


					// ---------Node hierarchy transform--------
					hipDeviceSynchronize();
					
					dim3 numBlocksNodeTransform((numVertices + numThreadsPerBlock.x - 1) / numThreadsPerBlock.x);
					_nodeMatrixTransform << <numBlocksNodeTransform, numThreadsPerBlock >> > (
						numVertices,
						dev_position,
						dev_normal,
						matrix,
						matrixNormal);

					checkCUDAError("Node hierarchy transformation");

					// at the end of the for loop of primitive
					// push dev pointers to map
					primitiveVector.push_back(PrimitiveDevBufPointers{
						primitive.mode,
						primitiveType,
						numPrimitives,
						numIndices,
						numVertices,

						dev_indices,
						dev_position,
						dev_normal,
						dev_texcoord0,

						dev_diffuseTex,
						diffuseTexWidth,
						diffuseTexHeight,

						dev_vertexOut	//VertexOut
					});

					totalNumPrimitives += numPrimitives;

				} // for each primitive

			} // for each mesh

		} // for each node

	}
	

	// 3. Malloc for dev_primitives
	{
		hipMalloc(&dev_primitives, totalNumPrimitives * sizeof(Primitive));
	}
	

	// Finally, hipFree raw dev_bufferViews
	{

		std::map<std::string, BufferByte*>::const_iterator it(bufferViewDevPointers.begin());
		std::map<std::string, BufferByte*>::const_iterator itEnd(bufferViewDevPointers.end());
			
			//bufferViewDevPointers

		for (; it != itEnd; it++) {
			hipFree(it->second);
		}

		checkCUDAError("Free BufferView Device Mem");
	}


}



__global__ 
void _vertexTransformAndAssembly(
	int numVertices, 
	PrimitiveDevBufPointers primitive, 
	glm::mat4 MVP, glm::mat4 MV, glm::mat3 MV_normal, 
	int width, int height) {

	// vertex id
	int vid = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (vid < numVertices) {

		// TODO: Apply vertex transformation here
		// Multiply the MVP matrix for each vertex position, this will transform everything into clipping space
		// Then divide the pos by its w element to transform into NDC space
		// Finally transform x and y to viewport space
		VertexOut& vertex = primitive.dev_verticesOut[vid];
		VertexAttributePosition pos = primitive.dev_position[vid];
		vertex.pos = MVP * glm::vec4(pos, 1);
		vertex.pos /= vertex.pos.w * 2.0f;
		vertex.pos.x = (vertex.pos.x + 0.5f) * width;
		vertex.pos.y = (0.5f - vertex.pos.y) * height;

		// TODO: Apply vertex assembly here
		// Assemble all attribute arrays into the primitive array
		vertex.eyePos = glm::vec3(MV * glm::vec4(pos, 1));
		vertex.eyeNor = glm::normalize(MV_normal * primitive.dev_normal[vid]);

		if (primitive.dev_diffuseTex != NULL) {
			vertex.dev_diffuseTex = primitive.dev_diffuseTex;
			vertex.texcoord0 = primitive.dev_texcoord0[vid];
			vertex.texWidth = primitive.diffuseTexWidth;
			vertex.texHeight = primitive.diffuseTexHeight;
		} else {
			vertex.color = glm::vec3((float)vid / numVertices, (float)vid / numVertices, 1.0f - ((float)vid / numVertices));
		}

	}
}



static int curPrimitiveBeginId = 0;

__global__ 
void _primitiveAssembly(int numIndices, int curPrimitiveBeginId, Primitive* dev_primitives, PrimitiveDevBufPointers primitive) {

	// index id
	int iid = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (iid < numIndices) {

		// Primitive assembly for triangles
		int pid; // id for cur primitives vector
		if (primitive.primitiveMode == TINYGLTF_MODE_TRIANGLES) {
			pid = iid / (int)primitive.primitiveType;
			dev_primitives[pid + curPrimitiveBeginId].v[iid % (int)primitive.primitiveType]
				= primitive.dev_verticesOut[primitive.dev_indices[iid]];
		}

		// TODO: other primitive types (point, line)
	}
}

__global__
void scanline(int width, int height, int numPrimitives, Primitive* primitives, Fragment* fragments, int* depth, int* mutex) {
	const int id = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (id < numPrimitives) {
		Primitive& prim = primitives[id];
		const glm::vec3 tri[] = { glm::vec3(prim.v[0].pos), glm::vec3(prim.v[1].pos), glm::vec3(prim.v[2].pos) };

		// Get triangle bounding region
		AABB bounds = getAABBForTriangle(tri);
		bounds.min.x = glm::min(width - 1.0f, glm::max(bounds.min.x, 0.0f));
		bounds.min.y = glm::min(height - 1.0f, glm::max(bounds.min.y, 0.0f));
		bounds.max.x = glm::max(0.0f, glm::min(bounds.max.x, width - 1.0f));
		bounds.max.y = glm::max(0.0f, glm::min(bounds.max.y, height - 1.0f));

		for (int x = bounds.min.x; x <= bounds.max.x; ++x) {
			for (int y = bounds.min.y; y <= bounds.max.y; ++y) {
				glm::vec3 bary = calculateBarycentricCoordinate(tri, glm::vec2(x, y));
				if (isBarycentricCoordInBounds(bary)) {
					const int index = x + y * width;
					float z = getZAtCoordinate(bary, tri);
					int zdepth = z * INT_MAX;

					bool locked;
					do {
						locked = atomicCAS(&mutex[index], 0, 1);
						if (!locked) {
							// Critical section
							if (zdepth < depth[index]) {
								if (prim.v[0].dev_diffuseTex) {
									glm::vec2 UV = bary.x * prim.v[0].texcoord0 + bary.y * prim.v[1].texcoord0 + bary.z * prim.v[2].texcoord0;
									int u = UV.x * prim.v[0].texWidth;
									int v = UV.y * prim.v[0].texHeight;
#if BILINEAR
									glm::vec3 col00 = glm::vec3(
										prim.v[0].dev_diffuseTex[(u + v * prim.v[0].texWidth) * 3],
										prim.v[0].dev_diffuseTex[(u + v * prim.v[0].texWidth) * 3 + 1],
										prim.v[0].dev_diffuseTex[(u + v * prim.v[0].texWidth) * 3 + 2]);
									glm::vec3 col10 = glm::vec3(
										prim.v[0].dev_diffuseTex[(u + 1 + v * prim.v[0].texWidth) * 3],
										prim.v[0].dev_diffuseTex[(u + 1 + v * prim.v[0].texWidth) * 3 + 1],
										prim.v[0].dev_diffuseTex[(u + 1 + v * prim.v[0].texWidth) * 3 + 2]);
									glm::vec3 col01 = glm::vec3(
										prim.v[0].dev_diffuseTex[(u + (v + 1) * prim.v[0].texWidth) * 3],
										prim.v[0].dev_diffuseTex[(u + (v + 1) * prim.v[0].texWidth) * 3 + 1],
										prim.v[0].dev_diffuseTex[(u + (v + 1) * prim.v[0].texWidth) * 3 + 2]);
									glm::vec3 col11 = glm::vec3(
										prim.v[0].dev_diffuseTex[(u + 1 + (v + 1) * prim.v[0].texWidth) * 3],
										prim.v[0].dev_diffuseTex[(u + 1 + (v + 1) * prim.v[0].texWidth) * 3 + 1],
										prim.v[0].dev_diffuseTex[(u + 1 + (v + 1) * prim.v[0].texWidth) * 3 + 2]);

									glm::vec3 mix1 = glm::mix(col10, col00, UV.x);
									glm::vec3 mix2 = glm::mix(col11, col01, UV.x);
									fragments[index].color = glm::mix(mix2, mix1, UV.y) / 255.0f;
#else
									fragments[index].color = glm::vec3(
										prim.v[0].dev_diffuseTex[(u + v * prim.v[0].texWidth) * 3],
										prim.v[0].dev_diffuseTex[(u + v * prim.v[0].texWidth) * 3 + 1],
										prim.v[0].dev_diffuseTex[(u + v * prim.v[0].texWidth) * 3 + 2]) / 255.0f;
#endif // BILINEAR

								} else {
									fragments[index].color = glm::vec3(bary.x * prim.v[0].color +
										bary.y * glm::vec3(prim.v[1].color) + bary.z * glm::vec3(prim.v[2].color));
								}
								fragments[index].eyeNor = glm::vec3(bary.x * prim.v[0].eyeNor +
									bary.y * glm::vec3(prim.v[1].eyeNor) + bary.z * glm::vec3(prim.v[2].eyeNor));
								depth[index] = zdepth;
							}
							mutex[index] = 0;
						}
					} while (locked);

				}
			}
		}
	}
}

/**
 * Perform rasterization.
 */
void rasterize(uchar4* pbo, const glm::mat4& MVP, const glm::mat4& MV, const glm::mat3 MV_normal) {
    int sideLength2d = 8;
    dim3 blockSize2d(sideLength2d, sideLength2d);
    dim3 blockCount2d((width  - 1) / blockSize2d.x + 1,
		(height - 1) / blockSize2d.y + 1);

	// Execute your rasterization pipeline here
	// (See README for rasterization pipeline outline.)

	// Vertex Process & primitive assembly
	curPrimitiveBeginId = 0;
	dim3 numThreadsPerBlock(128);
	{
		auto it = mesh2PrimitivesMap.begin();
		auto itEnd = mesh2PrimitivesMap.end();

		for (; it != itEnd; ++it) {
			auto p = (it->second).begin();	// each primitive
			auto pEnd = (it->second).end();
			for (; p != pEnd; ++p) {
				dim3 numBlocksForVertices((p->numVertices + numThreadsPerBlock.x - 1) / numThreadsPerBlock.x);
				dim3 numBlocksForIndices((p->numIndices + numThreadsPerBlock.x - 1) / numThreadsPerBlock.x);

				_vertexTransformAndAssembly << < numBlocksForVertices, numThreadsPerBlock >> >
					(p->numVertices, *p, MVP, MV, MV_normal, width, height);
				checkCUDAError("Vertex Processing");
				hipDeviceSynchronize();
				_primitiveAssembly << < numBlocksForIndices, numThreadsPerBlock >> >
					(p->numIndices, curPrimitiveBeginId, dev_primitives, *p);
				checkCUDAError("Primitive Assembly");

				curPrimitiveBeginId += p->numPrimitives;
			}
		}

		checkCUDAError("Vertex Processing and Primitive Assembly");
	}
	
	hipMemset(dev_fragmentBuffer, 0, width * height * sizeof(Fragment));
	initDepth << <blockCount2d, blockSize2d >> >(width, height, dev_depth);
	
	// TODO: rasterize
	dim3 numBlocksForPrimitives((curPrimitiveBeginId + numThreadsPerBlock.x - 1) / numThreadsPerBlock.x);
	scanline << <numBlocksForPrimitives, numThreadsPerBlock >> >
		(width, height, curPrimitiveBeginId, dev_primitives, dev_fragmentBuffer, dev_depth, mutex);

    // Copy depthbuffer colors into framebuffer
	int w = width / SSAA;
	int h = height / SSAA;
	dim3 blockCountShrink2d((w - 1) / blockSize2d.x + 1,
		(h - 1) / blockSize2d.y + 1);
	render << <blockCountShrink2d, blockSize2d >> >(w, h, dev_fragmentBuffer, dev_framebuffer);
	checkCUDAError("fragment shader");
    // Copy framebuffer into OpenGL buffer for OpenGL previewing
    sendImageToPBO<<<blockCountShrink2d, blockSize2d>>>(pbo, w, h, dev_framebuffer);
    checkCUDAError("copy render result to pbo");
}

/**
 * Called once at the end of the program to free CUDA memory.
 */
void rasterizeFree() {

    // deconstruct primitives attribute/indices device buffer

	auto it(mesh2PrimitivesMap.begin());
	auto itEnd(mesh2PrimitivesMap.end());
	for (; it != itEnd; ++it) {
		for (auto p = it->second.begin(); p != it->second.end(); ++p) {
			hipFree(p->dev_indices);
			hipFree(p->dev_position);
			hipFree(p->dev_normal);
			hipFree(p->dev_texcoord0);
			hipFree(p->dev_diffuseTex);

			hipFree(p->dev_verticesOut);

			
			//TODO: release other attributes and materials
		}
	}

	////////////

    hipFree(dev_primitives);
    dev_primitives = NULL;

	hipFree(dev_fragmentBuffer);
	dev_fragmentBuffer = NULL;

    hipFree(dev_framebuffer);
    dev_framebuffer = NULL;

	hipFree(dev_depth);
	dev_depth = NULL;

	hipFree(mutex);
	mutex = NULL;

    checkCUDAError("rasterize Free");
}
