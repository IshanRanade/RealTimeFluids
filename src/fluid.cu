#include "hip/hip_runtime.h"
#include "fluid.h"

#include <hip/hip_runtime.h>
#include <>


__global__ void fillGridCells(int n, GridCell *cells) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;

	if (index < n) {
		GridCell &cell = cells[index];
		cell.worldPosition = glm::vec3(0, 0, 0);
	}
}

void initSim() {
	hipMalloc(&dev_gridCells, NUM_CELLS * sizeof(GridCell));
	hipMemset(dev_gridCells, 0, NUM_CELLS * sizeof(GridCell));

	int blocks = (NUM_CELLS + blockSize - 1) / blockSize;
}