#include "hip/hip_runtime.h"
#include "fluid.h"
//#include "hierarchy.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <hipsolver.h>
#include <>
#include <iostream>
#include <thrust/random.h>
#include <glm/gtc/matrix_transform.hpp>
#include <algorithm>


#define ERRORCHECK 1
#define FILENAME (strrchr(__FILE__, '/') ? strrchr(__FILE__, '/') + 1 : __FILE__)
#define checkCUDAError(msg) checkCUDAErrorFn2(msg, FILENAME, __LINE__)
void checkCUDAErrorFn2(const char *msg, const char *file, int line) {
#if ERRORCHECK
	hipDeviceSynchronize();
	hipError_t err = hipGetLastError();
	if (hipSuccess == err) {
		return;
	}

	fprintf(stderr, "CUDA error");
	if (file) {
		fprintf(stderr, " (%s:%d)", file, line);
	}
	fprintf(stderr, ": %s: %s\n", msg, hipGetErrorString(err));
#  ifdef _WIN32
	getchar();
#  endif
	exit(EXIT_FAILURE);
#endif
}

__device__ int getCellCompressedIndex(int x, int y, int z) {
	return z * GRID_X * GRID_Y + y * GRID_X + x;
}

__device__ glm::vec3 getCellUncompressedCoordinates(int index) {
    const int z = index / (GRID_X * GRID_Y);
	index -= (z * GRID_X * GRID_Y);
    const int y = index / GRID_X;
    const int x = index % GRID_X;

	return glm::vec3(x, y, z);
}

__global__ void fillVBOData(int n, void *vbo, MarkerParticle *particles) {
    const int index = blockIdx.x * blockDim.x + threadIdx.x;

	float *vboFloat = (float*)vbo;

	if (index < n) {
		MarkerParticle &particle = particles[index];

		// Set the position
		vboFloat[6 * index + 0] = particle.worldPosition.x;
		vboFloat[6 * index + 1] = particle.worldPosition.y;
		vboFloat[6 * index + 2] = particle.worldPosition.z;

		// Set the color
		vboFloat[6 * index + 3] = particle.color.x;
		vboFloat[6 * index + 4] = particle.color.y;
		vboFloat[6 * index + 5] = particle.color.z;
	}
}

// Quadratic solver from scratchapixel.com
__device__ bool solveQuadratic(const float &a, const float &b, const float &c, float &x0, float &x1) {
    float discr = b * b - 4 * a * c;
    if (discr < 0) return false;
    else if (discr == 0) x0 = x1 = -0.5 * b / a;
    else {
        float q = (b > 0) ?
            -0.5 * (b + sqrt(discr)) :
            -0.5 * (b - sqrt(discr));
        x0 = q / a;
        x1 = c / q;
    }
    if (x0 > x1) {
        float temp = x0;
        x0 = x1;
        x1 = temp;
    }

    return true;
}

// Ray-Sphere Intersection from scratchapixel.com
__device__ float raySphereIntersect(glm::vec3 rayPos, glm::vec3 rayDir, glm::vec3 center, float radius2) {
    float t0, t1; // solutions for t if the ray intersects
    
    // analytic solution
    glm::vec3 L = rayPos - center;
    float a = glm::dot(rayDir, rayDir);
    float b = 2 * glm::dot(rayDir, L);
    float c = glm::dot(L, L) - radius2;
    if (!solveQuadratic(a, b, c, t0, t1)) return -1.0f;

    if (t0 > t1) {
        float temp = t0;
        t0 = t1;
        t1 = temp;
    }

    if (t0 < 0) {
        t0 = t1; // if t0 is negative, let's use t1 instead 
        if (t0 < 0) return -1.0f; // both t0 and t1 are negative 
    }

    return t0;
}

__device__ float smin(float a, float b, float k) {
    const float h = glm::clamp(0.5f + 0.5f * (b - a) / k, 0.0f, 1.0f);
    return glm::mix(b, a, h) - k * h * (1.0f - h);
}

__device__ glm::vec4 smin(glm::vec3 vecA, glm::vec3 vecB, float a, float b, float k) {
    const float h = glm::clamp(0.5f + 0.5f * (b - a) / k, 0.0f, 1.0f);
    return glm::vec4(glm::mix(vecA, vecB, h), glm::mix(b, a, h) - k * h * (1.0f - h));
}

__device__ bool inBounds(float value, float bounds) {
	return (value >= -bounds) && (value <= bounds);
}

__global__ void raycastPBO(int numParticles, uchar4 *pbo, MarkerParticle *particles, glm::vec3 camPos, Camera camera) {
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int idy = blockIdx.y * blockDim.y + threadIdx.y;

	if (idx < camera.resolution.x && idy < camera.resolution.y) {
        // Setup variables
        bool intersected = false;
        glm::vec3 rayPos = camPos;
        float distance = 1000.0f;
        const glm::vec3 view = camera.view;
        const glm::vec3 up = camera.up;
        const glm::vec3 right = camera.right;
        glm::vec3 normal = glm::vec3(0, 1, 0);

        const float yscaled = glm::tan(camera.fov.y * (3.1415927f / 180.0f));
        const float xscaled = (yscaled *  camera.resolution.x) / camera.resolution.y;
		glm::vec2 pixelLength = glm::vec2(2 * xscaled / camera.resolution.x, 2 * yscaled / camera.resolution.y);

		glm::vec3 rayDir = glm::normalize(view
			- right * pixelLength.x * ((float)idx - camera.resolution.x * 0.5f)
			- up * pixelLength.y * ((float)idy - camera.resolution.y * 0.5f)
		);

#if RAY_CAST
        // Ray-Sphere Intersection with all particles
        for (int i = 0; i < numParticles; ++i) {
            MarkerParticle& particle = particles[i];

            float t = raySphereIntersect(rayPos, rayDir, particle.worldPosition, PARTICLE_RADIUS);
            if (t > 0 && t < distance) {
                intersected = true;
                distance = t;
                normal = glm::normalize(rayPos + (rayDir * t) - particle.worldPosition);
                
            }
        }
        rayPos += rayDir * distance;

#elif SPHERE_MARCH
        int iterations = 0;
        const int maxIterations = 16;
        const float radius = 1.25f;

        // Sphere march for smoothed min marker particle
		while(distance > radius && iterations < maxIterations) {
			for(int i = 0; i < numParticles; ++i) {
				MarkerParticle& particle = particles[i];
                //distance = glm::min(distance, glm::distance(rayPos, particle.worldPosition));
				distance = smin(distance, glm::distance(rayPos, particle.worldPosition), PARTICLE_RADIUS);
				if(distance < radius) {
                    normal = glm::normalize(rayPos - particle.worldPosition);
					break;
				}
			}
			rayPos += rayDir * distance;
			++iterations;
		}
        intersected = distance < radius;
#endif

        const int index = idx + idy * camera.resolution.x;

		// Set the color
		if(intersected) {
            // Ray hit a marker particle
            glm::vec3 color = glm::vec3(50.f, 50.f, 255.f);
			float depth = glm::clamp(glm::distance(rayPos, camPos) / 10.0f, 0.0f, 1.0f);
            glm::vec3 lightPos = glm::vec3(2, 1, 0);
            float specularIntensity = 10.0f;

            glm::vec3 refl = glm::normalize(glm::normalize(camPos - rayPos) + glm::normalize(lightPos));
            float specularTerm = glm::pow(glm::max(glm::dot(refl, normal), 0.0f), specularIntensity);

            color = color * (depth + specularTerm);
			pbo[index].x = glm::min(color.x, 255.0f);
			pbo[index].y = glm::min(color.y, 255.0f);
			pbo[index].z = glm::min(color.z, 255.0f);
			pbo[index].w = 0;
		}
        else {
            // Clear background
            pbo[index].x = 205.0f;
            pbo[index].y = 205.0f;
            pbo[index].z = 240.0f;
            pbo[index].w = 0;
        }
	}
}

void raycastPBO(uchar4* pbo, glm::vec3 camPos, Camera camera) {
	/*
	// Initialize 3D quad tree hierarchy
	TreeNode* root = buildTree(std::vector<MarkerParticle> particles, int currentDepth, glm::vec3 boundMin, glm::vec3 boundMax);
	int numNodes = tree::treeSize(root);
	std::vector<LinearNode> flatTree;
	for (int i = 0; i < n; i++) {
		flatTree.push_back(LinearNode());
	}
	int offset = 0;
	flattenTree(root, sortedGeoms, flatTree, &offset);
	deleteTree(root);
	*/

	const dim3 BLOCK_SIZE2d(8, 8);
	const dim3 blocksPerGrid2d(
		(camera.resolution.x + BLOCK_SIZE2d.x - 1) / BLOCK_SIZE2d.x,
		(camera.resolution.y + BLOCK_SIZE2d.y - 1) / BLOCK_SIZE2d.y);
	raycastPBO<<<blocksPerGrid2d, BLOCK_SIZE2d >>>(NUM_MARKER_PARTICLES, pbo, dev_markerParticles, camPos, camera);
	checkCUDAError("raymarch to form PBO failed");
	hipDeviceSynchronize();	
}

__global__ void initializeGridCells(int n, GridCell *cells) {
    const int index = blockIdx.x * blockDim.x + threadIdx.x;

	if (index < n) {
		int *x;
		int *y;
		int *z;
		glm::vec3 coords = getCellUncompressedCoordinates(index);

		GridCell &cell = cells[index];
		cell.worldPosition = glm::vec3(coords.x * CELL_WIDTH, coords.y * CELL_WIDTH, coords.z * CELL_WIDTH);
	}
}

__global__ void setAllGridCellsToAir(int n, GridCell *cells) {
    const int index = blockIdx.x * blockDim.x + threadIdx.x;

	if (index < n) {
		GridCell &cell = cells[index];
		cell.cellType = AIR;
	}
}

__global__ void setGridCellsWithMarkerParticleToFluid(int n, GridCell *cells, MarkerParticle *particles) {
    const int index = blockIdx.x * blockDim.x + threadIdx.x;

	if (index < n) {
		MarkerParticle &particle = particles[index];

		int compressedCellIndex = getCellCompressedIndex((int)particle.worldPosition.x / CELL_WIDTH, (int)particle.worldPosition.x / CELL_WIDTH, (int)particle.worldPosition.x / CELL_WIDTH);
		
		GridCell &cell = cells[compressedCellIndex];
		cell.cellType = FLUID;
	}
}

void fillVBOsWithMarkerParticles(void *vbo) {
    const int blocks = (NUM_MARKER_PARTICLES + BLOCK_SIZE - 1) / BLOCK_SIZE;
	fillVBOData<<<blocks, BLOCK_SIZE>>>(NUM_MARKER_PARTICLES, vbo, dev_markerParticles);
	checkCUDAError("filling VBOs with marker particle data failed");
	hipDeviceSynchronize();
}

__global__ void generateRandomWorldPositionsForParticles(int n, MarkerParticle *particles) {
    const int index = blockIdx.x * blockDim.x + threadIdx.x;

	if (index < n) {
		// TODO: Fix these random generators
		thrust::default_random_engine rngX = thrust::default_random_engine(index | (index << 22));
		thrust::default_random_engine rngY = thrust::default_random_engine(index | (index << 15) ^ index);
		thrust::default_random_engine rngZ = thrust::default_random_engine(index ^ (index * 13));
		thrust::uniform_real_distribution<float> u01(0, 1);

		MarkerParticle &particle = particles[index];
		particle.worldPosition.x = 1.0 * u01(rngX) * GRID_X * CELL_WIDTH;
		particle.worldPosition.y = 1.0 * u01(rngX) * GRID_Y * CELL_WIDTH;
		particle.worldPosition.z = 1.0 * u01(rngX) * GRID_Z * CELL_WIDTH;

		particle.color = glm::vec3(0.2, 0.2, 1);
	}
}

//__device__ float getInterpolatedValue(float x, float y, float z, int componentIndex) {
//
//}
//
//__device__ glm::vec3 getVelocity(float x, float y, float z) {
//
//}
//
//__device__ glm::vec3 traceParticle(float x, float y, float z, float t) {
//
//}

__global__ void backwardsParticleTrace(int n, GridCell *cells) {
    const int index = blockIdx.x * blockDim.x + threadIdx.x;

	if (index < n) {
		GridCell &cell = cells[index];

		// For now just use simple Euler
		const glm::vec3 cellPosition = (getCellUncompressedCoordinates(index) * CELL_WIDTH) + glm::vec3(CELL_WIDTH / 2.0, CELL_WIDTH / 2.0, CELL_WIDTH / 2.0);
        const glm::vec3 oldPosition = cellPosition - TIME_STEP * cell.velocity;

		int prevCellIndex = getCellCompressedIndex((int)oldPosition.x, (int)oldPosition.y, (int)oldPosition.z);
		if (prevCellIndex < 0 || prevCellIndex >= GRID_X * GRID_Y * GRID_Z) {
			return;
		}

		GridCell &otherCell = cells[prevCellIndex];
		cell.tempVelocity = otherCell.velocity;
	}
}

__global__ void applyExternalForcesToGridCells(int n, GridCell *cells) {
    const int index = blockIdx.x * blockDim.x + threadIdx.x;

	if (index < n) {
		GridCell &cell = cells[index];

		// Apply gravity
		cell.tempVelocity = cell.velocity + glm::vec3(0, -9.8 * TIME_STEP, 0);
	}
}
__global__ void moveMarkerParticlesThroughField(int n, GridCell *cells, MarkerParticle *particles) {
    const int index = blockIdx.x * blockDim.x + threadIdx.x;

	if (index < n) {
		MarkerParticle &particle = particles[index];

		// Find the cell that this particle is in
		int cellIndex = getCellCompressedIndex(particle.worldPosition.x, particle.worldPosition.y, particle.worldPosition.z);
		GridCell &cell = cells[cellIndex];
		particle.worldPosition += TIME_STEP * cell.velocity;

		particle.worldPosition.x = glm::clamp(particle.worldPosition.x, 0.0f, GRID_X * CELL_WIDTH - 0.01f);
		particle.worldPosition.y = glm::clamp(particle.worldPosition.y, 0.0f, GRID_Y * CELL_WIDTH - 0.01f);
		particle.worldPosition.z = glm::clamp(particle.worldPosition.z, 0.0f, GRID_Z * CELL_WIDTH - 0.01f);
	}
}

__global__ void applyViscosity(int n, GridCell *cells) {
    const int index = blockIdx.x * blockDim.x + threadIdx.x;

	if (index < n) {
		GridCell &cell = cells[index];

		glm::vec3 cellCoords = getCellUncompressedCoordinates(index);

		int cellTopIndex    = getCellCompressedIndex(cellCoords.x, cellCoords.y + 1, cellCoords.z);
		int cellBottomIndex = getCellCompressedIndex(cellCoords.x, cellCoords.y - 1, cellCoords.z);
		int cellLeftIndex   = getCellCompressedIndex(cellCoords.x - 1, cellCoords.y, cellCoords.z);
		int cellRightIndex  = getCellCompressedIndex(cellCoords.x + 1, cellCoords.y, cellCoords.z);
		int cellFrontIndex  = getCellCompressedIndex(cellCoords.x, cellCoords.y, cellCoords.z + 1);
		int cellBackIndex   = getCellCompressedIndex(cellCoords.x, cellCoords.y, cellCoords.z - 1);

		float indices[6] = { cellTopIndex, cellBottomIndex, cellLeftIndex, cellRightIndex, cellFrontIndex, cellBackIndex };

		float laplacianX = 0.0;
		float laplacianY = 0.0;
		float laplacianZ = 0.0;
		for (int i = 0; i < 6; ++i) {
			int currCellIndex = indices[i];
			if (currCellIndex >= 0 && currCellIndex < GRID_X * GRID_Y * GRID_Z) {
				laplacianX += cells[currCellIndex].velocity.x;
				laplacianY += cells[currCellIndex].velocity.y;
				laplacianZ += cells[currCellIndex].velocity.z;
			}
		}

		laplacianX -= 6 * cell.velocity.x;
		laplacianY -= 6 * cell.velocity.y;
		laplacianZ -= 6 * cell.velocity.z;

		cell.tempVelocity = cell.velocity + TIME_STEP * VISCOSITY * glm::vec3(laplacianX, laplacianY, laplacianZ);
	}
}

__global__ void swapCellVelocities(int n, GridCell *cells) {
    const int index = blockIdx.x * blockDim.x + threadIdx.x;

	if (index < n) {
		GridCell &cell = cells[index];
		cell.velocity = cell.tempVelocity;
	}
}

__global__ void setupPressureCalc(int numCells, float* csrValA, int* csrRowPtrA, int* csrColIndA, float* vecB, GridCell* cells) {
    const int index = blockIdx.x * blockDim.x + threadIdx.x;

	if (index > 10 && index < 12) {
		glm::vec3 gridPos = getCellUncompressedCoordinates(index);

		// Starting index of current row
		csrRowPtrA[index] = index * 27;

		int nonSolid = -26;
		float airCells = 0.0f;
		for (int i = 0; i < 26; ++i) {
			int x = i % 3 - 1;
			int y = (i / 3) % 3 - 1;
			int z = i / 9 - 1;
			int adjacent = index + x + (y * GRID_X) + (z * GRID_X * GRID_Y);
			if (gridPos.x + x < 0 || gridPos.x + x >= GRID_X || gridPos.y + y < 0 || gridPos.y + y >= GRID_Y || gridPos.x + x < 0 || gridPos.z + z >= GRID_Z) {
				csrColIndA[index * 27 + i] = 0;
				csrRowPtrA[index * 27 + i] = 0;
				++nonSolid;
				continue;
			}
			GridCell cell = cells[adjacent];

			// Set index of adjacent cell
			csrColIndA[index * 27 + i] = adjacent;

			// Set value of matrix element
			csrRowPtrA[index * 27 + i] = cell.cellType == FLUID ? 1.0f : 0.0f;
			airCells += cell.cellType == AIR ? 1.0f : 0.0f;
		}
		// Set matrix value for current grid cell
		csrColIndA[index * 27 + 26] = nonSolid;
		csrRowPtrA[index * 27 + 26] = index;

		// Set value of b vector for pressure linear solver
		float divU = 0.0f;
		if (gridPos.x - 1 > 0) {
			divU += cells[index - 1].velocity.x - cells[index].velocity.x;
		}
		if (gridPos.y - 1 > 0) {
			divU += cells[index - GRID_X].velocity.x - cells[index].velocity.x;
		}
		if (gridPos.z - 1 > 0) {
			divU += cells[index - GRID_X * GRID_Y].velocity.x - cells[index].velocity.x;
		}
		vecB[index] = (WIDTH_DIV_TIME) * divU - airCells;
	}
}

__global__ void copyPressureToCells(int numCells, float* vecX, GridCell* cells) {
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index < numCells) {
		//printf("%d: %f\n", index, vecX[index]);
		cells[index].pressure = vecX[index];
	}
}

void initHierarchicalPressureGrids() {
    // Calculate number of grid levels`
    GRID_LEVELS = std::floor(log2(std::min(std::min(GRID_X, GRID_Y), GRID_Z)));

    // Allocate space for primary grid cells
    hipMalloc(&dev_gridCells, NUM_CELLS * sizeof(GridCell));

    // Create grid array and primary grid
    grids = new Grid[GRID_LEVELS];
    grids[0].setGrid(0, GRID_X, GRID_Y, GRID_Z);
    grids[0].dev_cells = dev_gridCells;

    for(int d = 1; d < GRID_LEVELS; ++d) {
        // Create and allocate space for sub grid cells
        grids[d].setGrid(d, grids[d - 1].gridX / 2, grids[d - 1].gridY / 2, grids[d - 1].gridZ / 2);
        hipMalloc(&grids[d].dev_cells, grids[d].numCells * sizeof(GridCell));
    }
}

void initSim() {
    // Init hierarchical pressure grids
    initHierarchicalPressureGrids();

	// Allocate space for all of the marker particles
	hipMalloc(&dev_markerParticles, NUM_MARKER_PARTICLES * sizeof(MarkerParticle));

	// Allocate space for sparse linear solver of pressures
	/*nnz = NUM_CELLS * 27;
	hipMalloc(&csrValA, nnz * sizeof(float));
	hipMalloc(&csrRowPtrA, NUM_CELLS * sizeof(int));
	hipMalloc(&csrColIndA, nnz * sizeof(int));
	hipMalloc(&vecX, NUM_CELLS * sizeof(float));
	hipMalloc(&vecB, NUM_CELLS * sizeof(float));*/

	// Create random world positions for all of the particles
	generateRandomWorldPositionsForParticles<<<BLOCKS_PARTICLES, BLOCK_SIZE>>>(NUM_MARKER_PARTICLES, dev_markerParticles);
	checkCUDAError("generating initial world positions for marker particles failed");
	hipDeviceSynchronize();

	// Initialize the grid cells
	initializeGridCells<<<BLOCKS_CELLS, BLOCK_SIZE>>>(NUM_CELLS, dev_gridCells);
	checkCUDAError("initializing the grid cells failed");
	hipDeviceSynchronize();
}

void iterateSim() {
	// Make all the cells temporarily air cells
	setAllGridCellsToAir<<<BLOCKS_CELLS, BLOCK_SIZE>>>(NUM_CELLS, dev_gridCells);
	checkCUDAError("marking all cells as air cells failed");
	hipDeviceSynchronize();

	// Mark all cells with a marker particle as a fluid cell
	setGridCellsWithMarkerParticleToFluid<<<BLOCKS_PARTICLES, BLOCK_SIZE>>>(NUM_MARKER_PARTICLES, dev_gridCells, dev_markerParticles);
	checkCUDAError("marking all cells with a marker particle as fluid cells failed");
	hipDeviceSynchronize();

	// Apply convection to velocities using a backwards particle trace
	backwardsParticleTrace<<<BLOCKS_CELLS, BLOCK_SIZE>>>(NUM_CELLS, dev_gridCells);
	checkCUDAError("convecting velocities using a backwards particle trace failed");
	hipDeviceSynchronize();

	// Set each cell velocity to be the temp velocity, needed since previous step had to save old velocities during calculations
	swapCellVelocities << <BLOCKS_CELLS, BLOCK_SIZE >> > (NUM_CELLS, dev_gridCells);
	checkCUDAError("swapping velocities in cells failed");
	hipDeviceSynchronize();

	// Apply external forces to grid cell velocities
	applyExternalForcesToGridCells<<<BLOCKS_CELLS, BLOCK_SIZE>>>(NUM_CELLS, dev_gridCells);
	checkCUDAError("applying external forces to cells failed");
	hipDeviceSynchronize();

	// Set each cell velocity to be the temp velocity, needed since previous step had to save old velocities during calculations
	swapCellVelocities << <BLOCKS_CELLS, BLOCK_SIZE >> > (NUM_CELLS, dev_gridCells);
	checkCUDAError("swapping velocities in cells failed");
	hipDeviceSynchronize();

	// Apply viscosity to velocities
	applyViscosity<<<BLOCKS_CELLS, BLOCK_SIZE>>>(NUM_CELLS, dev_gridCells);
	checkCUDAError("applying viscosity failed");
	hipDeviceSynchronize();

	// Set each cell velocity to be the temp velocity, needed since previous step had to save old velocities during calculations
	swapCellVelocities << <BLOCKS_CELLS, BLOCK_SIZE >> > (NUM_CELLS, dev_gridCells);
	checkCUDAError("swapping velocities in cells failed");
	hipDeviceSynchronize();

	// Calculate pressure
	/*setupPressureCalc << <BLOCKS_CELLS, BLOCK_SIZE >> > (NUM_CELLS, csrValA, csrRowPtrA, csrColIndA, vecB, dev_gridCells);
	checkCUDAError("setup pressure calc failed");
	hipDeviceSynchronize();

	hipsolverSpHandle_t cusolver_handle;
	hipsolverStatus_t cusolver_status;
	cusolver_status = hipsolverSpCreate(&cusolver_handle);
	//std::cout << "status create cusolver handle: " << cusolver_status << std::endl;
	int singularity = 0;
	hipsparseMatDescr_t descrA;
	hipsparseCreateMatDescr(&descrA);
	cusolver_status = hipsolverSpScsrlsvqr(cusolver_handle, NUM_CELLS, nnz, descrA, csrValA, csrRowPtrA, csrColIndA, vecB, 1e-5, 0, vecX, &singularity);

	copyPressureToCells << <BLOCKS_CELLS, BLOCK_SIZE >> > (NUM_CELLS, vecX, dev_gridCells);
	checkCUDAError("copy pressure to cells failed");
	hipDeviceSynchronize();*/

	// Apply pressure

	// Extrapolate fluid velocities into surrounding cells

	// Set the velocities of surrounding cells

	// Move the marker particles through the velocity field
	moveMarkerParticlesThroughField<<<BLOCKS_PARTICLES, BLOCK_SIZE>>>(NUM_MARKER_PARTICLES, dev_gridCells, dev_markerParticles);
	checkCUDAError("moving marker particles through velocity field failed");
	hipDeviceSynchronize();
}
